
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#define N 64
#define TPB 32


float scale(int i, int n)
{
 return ((float) i)/((float) (n-1));
}

__device__ float distance(float x1, float x2)
{
 float x = (x1-x2)*(x1-x2);
 return sqrt(x);
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref)
{
 const int i = blockIdx.x*blockDim.x + threadIdx.x;
 const float x = d_in[i];
 d_out[i] = distance(x,ref);
 printf("out [ %d ] = %f \n", i, d_out[i]);
}

int main()
{
 float *in = new float;
 float *out = new float;
 const float ref = 0.5;  

 hipMallocManaged(&in, N*sizeof(float));
 hipMallocManaged(&out, N*sizeof(float));
 
 for (int i=0; i<N; i++)
 {
  in[i] = scale(i,N);
 }

 distanceKernel<<<N/TPB,TPB>>>(out, in, ref);
 hipDeviceSynchronize();

 hipFree(in);
 hipFree(out);  
}
