#include "hip/hip_runtime.h"
#include <iostream>
#include "kernel.h"
#define TPB 64
#define VERSION 2

__global__ void dotKernel(float *d_res, const float *d_a, const float *d_b, const int n)
{
 const int idx = threadIdx.x + blockIdx.x*blockDim.x;
 if(idx>n) return;

 __shared__ float s_prod[TPB];
 const int s_idx = threadIdx.x;

 s_prod[s_idx] = d_a[idx]*d_b[idx];
 __syncthreads();


 if (VERSION == 1)
 { 
  if (s_idx == 0)
  {
   float blockSum = 0;
   for (int j=0; j<blockDim.x; j++)
   {
    blockSum += s_prod[j];
   }
   atomicAdd(d_res,blockSum);
  }
 }
 else if (VERSION == 2)
 {

  // reduction
  for (int i = blockDim.x/2; i>0; i /= 2)
  {
   if (s_idx < i) s_prod[s_idx] += s_prod[s_idx + i];
   __syncthreads();
  }
  
  if(s_idx ==0) atomicAdd(d_res,s_prod[s_idx]);
 }
 

}


void dot(float *res, const float *a, const float *b, const int n)
{
 float *d_res;
 float *d_a = 0;
 float *d_b = 0;

 hipMalloc(&d_res,sizeof(float));
 hipMalloc(&d_a,n*sizeof(float));
 hipMalloc(&d_b,n*sizeof(float));

 hipMemset(d_res,0,sizeof(float));
 hipMemcpy(d_a,a,n*sizeof(float),hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,n*sizeof(float),hipMemcpyHostToDevice);

 dotKernel<<<(n+TPB-1)/TPB,TPB>>>(d_res, d_a, d_b, n);
 hipMemcpy(res,d_res,sizeof(float),hipMemcpyDeviceToHost);

 hipFree(d_res);
 hipFree(d_a);
 hipFree(d_b);
}
