
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
using namespace std;

#define W 500
#define H 500
#define TPB 32

__device__ float square(float x)
{
 return (x*x);
}

__global__ void distKernel(float *dout, int w, int h, float2 pos)
{
 const int c = blockIdx.x*blockDim.x + threadIdx.x;
 const int r = blockIdx.y*blockDim.y + threadIdx.y;
 const int i = r*w + c;
 if ((c >= w) || (r >= h)) return;
 dout[i] = sqrt(square(c-pos.x) + square(r-pos.y));
}

int main()
{
 float *out = new float [W*H]; 
 float *dout = new float;
 const int size = W*H*sizeof(float);
 hipMalloc(&dout,size); 
 const float2 pos = {0.0, 0.0};
 const dim3 tpb(TPB, TPB);
 const dim3 bpg((W+TPB-1)/TPB, (H+TPB-1)/TPB);
 
 distKernel<<<bpg,tpb>>>(dout,W,H,pos);
 hipMemcpy(out,dout,size,hipMemcpyDeviceToHost);

 hipFree(dout);
 delete [] out;
}

