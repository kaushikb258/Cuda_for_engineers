
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
using namespace std;

#define W 500
#define H 500
#define TPB 32

__device__ unsigned char clip(int n)
{
 if (n>255) return n;
 else if (n<0) return 0;
 else return n;
}

__device__ int square(int x)
{
 return (x*x);
}

__global__ void distKernel(uchar4 *dout, int w, int h, int2 pos)
{
 const int c = blockIdx.x*blockDim.x + threadIdx.x;
 const int r = blockIdx.y*blockDim.y + threadIdx.y;
 const int i = r*w + c;
 if ((c >= w) || (r >= h)) return;
 int d = sqrtf(square(c-pos.x) + square(r-pos.y));
 unsigned char intensity = clip(255-d);

 dout[i].x = intensity; // red
 dout[i].y = intensity; // green
 dout[i].z = 0; // blue
 dout[i].w = 255; // opaque
}

int main()
{
 uchar4 *out = new uchar4 [W*H]; 
 uchar4 *dout = new uchar4;
 const int size = W*H*sizeof(uchar4);
 hipMalloc(&dout,size); 
 const int2 pos = {0, 0};
 const dim3 tpb(TPB, TPB);
 const dim3 bpg((W+TPB-1)/TPB, (H+TPB-1)/TPB);
 
 distKernel<<<bpg,tpb>>>(dout,W,H,pos);
 hipMemcpy(out,dout,size,hipMemcpyDeviceToHost);

 hipFree(dout);
 delete [] out;
}

