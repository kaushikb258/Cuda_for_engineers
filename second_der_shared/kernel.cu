#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "kernel.h"
#include "global.h"
using namespace std;

__device__ float compDer(float xim1, float xi, float xip1)
{
 return ((xim1 - 2.0*xi + xip1)/(h*h));
}

__global__ void derivative(float  *d_in, float *d_out)
{
 const int idx = blockIdx.x*blockDim.x + threadIdx.x;
 if (idx>N) return;
 
 extern __shared__ float s_in[];
 
 // interior cells
 s_in[threadIdx.x+1] = d_in[idx]; 

 // ghost cells
 if(blockIdx.x != 0 && threadIdx.x==0) s_in[0] = d_in[idx-1];  
 if(blockIdx.x != gridDim.x-1 && threadIdx.x==blockDim.x-1) s_in[blockDim.x+1] = d_in[idx+1]; 

 // boundary cells, use periodic BC
 if (blockIdx.x==0 && threadIdx.x==0) s_in[0] = d_in[N-1];  
 if (blockIdx.x==gridDim.x-1 && threadIdx.x==blockDim.x-1) s_in[blockDim.x+1] = d_in[0];

 __syncthreads();
 
 d_out[idx] = compDer(s_in[threadIdx.x],s_in[threadIdx.x+1],s_in[threadIdx.x+2]); 
}

void func(float *in, float *out)
{
 float *d_in = 0;
 float *d_out = 0;
 int size = N*sizeof(float);
 const int BPG = (N + TPB -1)/TPB;
 cout<<"BPG = "<<BPG<<endl;
 cout<<"TPB = "<<TPB<<endl;
 const int s_size = (TPB+2)*sizeof(float);

 hipMalloc(&d_in,size); 
 hipMalloc(&d_out,size);  

 hipMemcpy(d_in,in,size,hipMemcpyHostToDevice);
 derivative<<<BPG,TPB,s_size>>>(d_in,d_out);
 hipMemcpy(out,d_out,size,hipMemcpyDeviceToHost);

 hipFree(d_in);
 hipFree(d_out);
}
